#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <math.h>
#include <assert.h>

#include "test_utils.hh"

namespace testutils {

	void initializeTensorWithValue(Matrix M, float value) {
		for (int x = 0; x < M.shape.x; x++) {
			for (int y = 0; y < M.shape.y; y++) {
				M[y * M.shape.x + x] = value;
			}
		}
	}

	void initializeTensorRandomlyInRange(Matrix M, float min, float max) {
		srand( time(NULL) );
		for (int x = 0; x < M.shape.x; x++) {
			for (int y = 0; y < M.shape.y; y++) {
				M[y * M.shape.x + x] = (static_cast<float>(rand()) / RAND_MAX) * (max - min) + min;
			}
		}
	}

	void initializeEachTensorRowWithValue(Matrix M, std::vector<float> values) {
		assert(M.shape.y == values.size());

		for (int x = 0; x < M.shape.x; x++) {
			for (int y = 0; y < M.shape.y; y++) {
				M[y * M.shape.x + x] = values[y];
			}
		}
	}

	void initializeEachTensorColWithValue(Matrix M, std::vector<float> values) {
		assert(M.shape.x == values.size());

		for (int x = 0; x < M.shape.x; x++) {
			for (int y = 0; y < M.shape.y; y++) {
				M[y * M.shape.x + x] = values[x];
			}
		}
	}

	float sigmoid(float x) {
		return exp(x) / (1 + exp(x));
	}

}
